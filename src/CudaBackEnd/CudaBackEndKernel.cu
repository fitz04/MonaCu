#include "hip/hip_runtime.h"
//#include "MonaCu.cuh"

#include <hipblas.h>
#include <hipblas.h>
#include <hipblas.h>
#include <nvfunctional>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include "../../include/MonaCuTypeDef.h"

static int threadsPerBlock = 256;

template<typename Real>
__global__ void vecCopy(int numElements, const Real *src, Real *dst){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) dst[i] = src[i];
}

template<typename Real>
__global__ void vecAdd(int numElements, Real *A, Real *B, Real *C){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) C[i] = A[i] + B[i];
}

template<typename Real>
__global__ void vecMinus(int numElements, Real *A, Real *B, Real *C){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) C[i] = A[i] - B[i];
}

template<typename Real>
__global__ void vecMul(int numElements, Real *A, Real *B, Real *C){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) C[i] = A[i] * B[i];
}

template<typename Real>
__global__ void vecDiv(int numElements, Real *A, Real *B, Real *C){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) C[i] = A[i] / B[i];
}


int getCudaBlock(){                                                                              
    return threadsPerBlock;
}
                                                                                             
int setCudaBlock(int num){                                                                       
    threadsPerBlock = num;                                                                   
    return threadsPerBlock;                                                                  
}                                                                                            

void* vectorAlloc(int numElements, int typeSize){
    void *tmp;
    hipError_t res = hipMalloc((void**)&tmp, typeSize*numElements);
    if (res != hipSuccess) return NULL;
    return tmp;
}

MonaCu::error_t vectorFree(void *dst){
    hipError_t res = hipFree(dst);
    if (res != hipSuccess) return MonaCu::Fail;
    return MonaCu::Success;
}

#define FUNCTION_INSTANTIATION(TYPE)                                                            \
void vectorCopy(int numElements, const TYPE *src, TYPE *dst){                                   \
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;                  \
    vecCopy<TYPE> << <blocksPerGrid, threadsPerBlock >> >(numElements, src, dst);               \
}                                                                                               \
void vectorAdd(int numElements, TYPE *A, TYPE *B, TYPE *dst){                                   \
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;                  \
    vecAdd<TYPE> << <blocksPerGrid, threadsPerBlock >> >(numElements, A, B, dst);               \
}                                                                                               \
void vectorMinus(int numElements, TYPE *A, TYPE *B, TYPE *dst){                                 \
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;                  \
    vecMinus<TYPE> << <blocksPerGrid, threadsPerBlock >> >(numElements, A, B, dst);             \
}                                                                                               \
void vectorMul(int numElements, TYPE *A, TYPE *B, TYPE *dst){                                   \
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;                  \
    vecMul<TYPE> << <blocksPerGrid, threadsPerBlock >> >(numElements, A, B, dst);               \
}                                                                                               \
void vectorDiv(int numElements, TYPE *A, TYPE *B, TYPE *dst){                                   \
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;                  \
    vecDiv<TYPE> << <blocksPerGrid, threadsPerBlock >> >(numElements, A, B, dst);               \
}                                                                                               \
MonaCu::error_t getData(int numElements, const TYPE *src, TYPE *dst){                           \
    hipError_t res = hipMemcpy(dst, src, sizeof(TYPE)*numElements, hipMemcpyDeviceToHost);   \
    if (res != hipSuccess) return MonaCu::Fail;                                                \
    return MonaCu::Success;                                                                     \
}                                                                                               \
MonaCu::error_t setData(int numElements, const TYPE *src, TYPE *dst){                           \
    hipError_t res = hipMemcpy(dst, src, sizeof(TYPE)*numElements, hipMemcpyHostToDevice);   \
    if (res != hipSuccess) return MonaCu::Fail;                                                \
    return MonaCu::Success;                                                                     \
}

FUNCTION_INSTANTIATION(short);
FUNCTION_INSTANTIATION(int);
FUNCTION_INSTANTIATION(float);
FUNCTION_INSTANTIATION(double);